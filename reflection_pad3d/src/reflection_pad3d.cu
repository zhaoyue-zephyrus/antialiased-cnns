#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <THC/THCAtomics.cuh>

#include <thrust/pair.h>

__device__
inline thrust::pair<int64_t, int64_t> get_index_mapping3d(
    int64_t input_dim_x, int64_t input_dim_y, int64_t input_dim_z,
    int64_t output_dim_x, int64_t output_dim_y, int64_t output_dim_z,
    int64_t pad_f, int64_t pad_bk,
    int64_t pad_t, int64_t pad_b,
    int64_t pad_l, int64_t pad_r,
    int64_t output_xyz) {
  auto input_offset =
    (blockIdx.y + blockIdx.z * gridDim.y) * input_dim_x * input_dim_y * input_dim_z;
  auto output_offset =
    (blockIdx.y + blockIdx.z * gridDim.y) * output_dim_x * output_dim_y * output_dim_z;

  auto output_x = output_xyz % output_dim_x;
  auto output_y = output_xyz / output_dim_x % output_dim_y;
  auto output_z = output_xyz / (output_dim_x * output_dim_y);

  auto i_start_x = ::max(int64_t(0), -pad_l);
  auto i_start_y = ::max(int64_t(0), -pad_t);
  auto i_start_z = ::max(int64_t(0), -pad_f);
  auto o_start_x = ::max(int64_t(0), pad_l);
  auto o_start_y = ::max(int64_t(0), pad_t);
  auto o_start_z = ::max(int64_t(0), pad_f);
  

  auto input_x = ::abs(output_x - pad_l)
                 - ::abs(output_x - (input_dim_x + pad_l - 1))
                 - output_x
                 + 2 * pad_l + input_dim_x - 1
                 - o_start_x + i_start_x;
  auto input_y = ::abs(output_y - pad_t)
                 - ::abs(output_y - (input_dim_y + pad_t - 1))
                 - output_y
                 + 2 * pad_t + input_dim_y - 1
                 - o_start_y + i_start_y;
  auto input_z = ::abs(output_z - pad_f)
                 - ::abs(output_z - (input_dim_z + pad_f - 1))
                 - output_z
                 + 2 * pad_f + input_dim_z - 1
                 - o_start_z + i_start_z;
 
  return thrust::make_pair<int64_t, int64_t>(
    input_offset
    + input_z * input_dim_y * input_dim_x
    + input_y * input_dim_x + input_x,
    output_offset
    + output_z * output_dim_y * output_dim_x
    + output_y * output_dim_x + output_x
  );
}

template <typename scalar_t>
__global__ void reflection_pad3d_out_kernel(
    scalar_t *input, scalar_t *output,
    int64_t input_dim_x, int64_t input_dim_y,
    int64_t input_dim_z,
    int pad_f, int pad_bk,
    int pad_t, int pad_b,
    int pad_l, int pad_r) {
  auto output_xyz = threadIdx.x + blockIdx.x * blockDim.x;
  auto output_dim_x = input_dim_x + pad_l + pad_r;
  auto output_dim_y = input_dim_y + pad_t + pad_b;
  auto output_dim_z = input_dim_z + pad_f + pad_bk;

  if (output_xyz < output_dim_x * output_dim_y * output_dim_z) {
    auto index_pair = get_index_mapping3d(
      input_dim_x, input_dim_y, input_dim_z,
      output_dim_x, output_dim_y, output_dim_z,
      pad_f, pad_bk, pad_t, pad_b, pad_l, pad_r,
      output_xyz);

    output[index_pair.second] = input[index_pair.first];
  }
}

template <typename scalar_t>
__global__ void reflection_pad3d_backward_out_kernel(
    scalar_t *grad_input, scalar_t *grad_output,
    int64_t input_dim_x, int64_t input_dim_y,
    int64_t input_dim_z,
    int pad_f, int pad_bk,
    int pad_t, int pad_b,
    int pad_l, int pad_r) {
  auto output_xyz = threadIdx.x + blockIdx.x * blockDim.x;
  auto output_dim_x = input_dim_x + pad_l + pad_r;
  auto output_dim_y = input_dim_y + pad_t + pad_b;
  auto output_dim_z = input_dim_z + pad_f + pad_bk;

  if (output_xyz < output_dim_x * output_dim_y * output_dim_z) {
    auto index_pair = get_index_mapping3d(
      input_dim_x, input_dim_y, input_dim_z,
      output_dim_x, output_dim_y, output_dim_z,
      pad_f, pad_bk, pad_t, pad_b, pad_l, pad_r,
      output_xyz);

    atomicAdd(&grad_input[index_pair.first], grad_output[index_pair.second]);
  }
}

void reflection_pad3d_out_template(
     at::Tensor &output, at::Tensor &input_,
     int64_t pad_l, int64_t pad_r,
     int64_t pad_t, int64_t pad_b,
     int64_t pad_f, int64_t pad_bk) {
  int plane_dim = 0;
  int dim_t = 1;
  int dim_h = 2;
  int dim_w = 3;
  int nbatch = 1;

  if (input_.ndimension() == 5) {
    nbatch = input_.size(0);
    plane_dim++;
    dim_t++;
    dim_h++;
    dim_w++;
  }

  int nplane = input_.size(plane_dim);
  int input_t = input_.size(dim_t);
  int input_h = input_.size(dim_h);
  int input_w = input_.size(dim_w);

  if (pad_l >= input_w || pad_r >= input_w) {
    printf("Padding size should be less than the corresponding input dimension");
    return ;
  }
  if (pad_t >= input_h || pad_b >= input_h) {
    printf("Padding size should be less than the corresponding input dimension");
    return ;
  }
  if (pad_f >= input_t || pad_bk >= input_t) {
    printf("Padding size should be less than the corresponding input dimension");
    return ;  
  }

  int output_t = input_t + pad_f + pad_bk;
  int output_h = input_h + pad_t + pad_b;
  int output_w = input_w + pad_l + pad_r;

  if (input_.ndimension() == 4){
    output.resize_({nplane, output_t, output_h, output_w});
  } else {
    output.resize_({nbatch, nplane, output_t, output_h, output_w});
  }
  at::Tensor input = input_.contiguous();

  int output_plane_size = output_t * output_h * output_w;
  dim3 block_size(output_plane_size > 256 ? 256 : output_plane_size);
  dim3 grid_size(
    (int) std::ceil(output_plane_size / 256.0), nplane, nbatch);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    input.type(), "reflection_pad3d_out_template", [&]{
      reflection_pad3d_out_kernel<<<
        grid_size, block_size, 0, at::cuda::getCurrentCUDAStream()>>>(
          input.data<scalar_t>(), output.data<scalar_t>(),
          input_w, input_h, input_t,
          pad_f, pad_bk, pad_t, pad_b, pad_l, pad_r);
    }
  );

}

void reflection_pad3d_backward_out_template(
     at::Tensor &grad_input, at::Tensor &grad_output_,
     at::Tensor &input,
     int64_t pad_l, int64_t pad_r,
     int64_t pad_t, int64_t pad_b,
     int64_t pad_f, int64_t pad_bk) {
  int plane_dim = 0;
  int dim_t = 1;
  int dim_h = 2;
  int dim_w = 3;
  int nbatch = 1;

  if (input.ndimension() == 5) {
    nbatch = input.size(0);
    plane_dim++;
    dim_t++;
    dim_h++;
    dim_w++;
  }

  int nplane = input.size(plane_dim);
  int input_t = input.size(dim_t);
  int input_h = input.size(dim_h);
  int input_w = input.size(dim_w);

  int output_t = input_t + pad_f + pad_bk;
  int output_h = input_h + pad_t + pad_b;
  int output_w = input_w + pad_l + pad_r;

  if (output_w != grad_output_.size(dim_w)) {
    printf("grad_output width unexpected.");
    return ;
  }
  if (output_h != grad_output_.size(dim_h)) {
    printf("grad_output height unexpected.");
    return ;
  }
  if (output_t != grad_output_.size(dim_t)) {
    printf("grad_output depth unexpected.");
    return ;
  }

  at::Tensor grad_output = grad_output_.contiguous();

  int output_plane_size = output_t * output_h * output_w;
  dim3 block_size(output_plane_size > 256 ? 256 : output_plane_size);
  dim3 grid_size(
    (int) std::ceil(output_plane_size / 256.0), nplane, nbatch);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    input.type(), "reflection_pad3d_backward_out_template", [&]{
      reflection_pad3d_backward_out_kernel<<<
        grid_size, block_size, 0, at::cuda::getCurrentCUDAStream()>>>(
          grad_input.data<scalar_t>(), grad_output.data<scalar_t>(),
          input_w, input_h, input_t,
          pad_f, pad_bk, pad_t, pad_b, pad_l, pad_r);
    }
  );

}
